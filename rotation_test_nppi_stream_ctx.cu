#include <iostream>
#include <stdio.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "helper_functions.h"

#include <opencv2/opencv.hpp>

//#include <nppi.h>
#include <nppcore.h>
#include <nppi_geometry_transforms.h>

using namespace std;
using namespace cv;

Mat read_BMP_opencv(char* filename, int& w, int& h);

int main()
{
	int f_width, f_height;
	char buf[256];

	hipEvent_t start, stop;
	float  elapsedTime;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int img_num = 32; 
	const int stream_num = 4;
	int n_iter = 50; 

	hipStream_t stream[stream_num];
	NppStreamContext nppStreamContext[stream_num];
	hipDeviceProp_t oDeviceProperties;

	for (int n = 0; n < stream_num; n++)
	{
		hipStreamCreate(&stream[n]);
		nppStreamContext[n].nCudaDeviceId = 0;
		nppStreamContext[n].hStream = stream[n];

		hipDeviceGetAttribute(&nppStreamContext[n].nCudaDevAttrComputeCapabilityMajor,
			hipDeviceAttributeComputeCapabilityMajor,
			nppStreamContext[n].nCudaDeviceId);

		hipDeviceGetAttribute(&nppStreamContext[n].nCudaDevAttrComputeCapabilityMinor,
			hipDeviceAttributeComputeCapabilityMinor,
			nppStreamContext[n].nCudaDeviceId);

		hipStreamGetFlags(nppStreamContext[n].hStream, &nppStreamContext[n].nStreamFlags);

		hipGetDevice(&nppStreamContext[n].nCudaDeviceId);
		hipGetDeviceProperties(&oDeviceProperties, nppStreamContext[n].nCudaDeviceId);

		nppStreamContext[n].nMultiProcessorCount = oDeviceProperties.multiProcessorCount;
		nppStreamContext[n].nMaxThreadsPerMultiProcessor = oDeviceProperties.maxThreadsPerMultiProcessor;
		nppStreamContext[n].nMaxThreadsPerBlock = oDeviceProperties.maxThreadsPerBlock;
		nppStreamContext[n].nSharedMemPerBlock = oDeviceProperties.sharedMemPerBlock;
	}

	cv::Mat::setDefaultAllocator(cv::cuda::HostMem::getAllocator(cv::cuda::HostMem::AllocType::PAGE_LOCKED));

	double angle = 3.8;

	///////////////////////////*********************** Memory Allocation ***********************/////////////////////////
	Mat* img = new Mat[img_num];
	Mat* result = new Mat[img_num];

	Npp8u* *data = new Npp8u*[img_num];
	Npp8u* *d_data = new Npp8u*[img_num];
	Npp8u* *d_rotated = new Npp8u*[img_num];
	Npp8u* *h_result = new Npp8u*[img_num];



	for (int i = 0; i < img_num; i++)
	{
		img[i] = read_BMP_opencv("input_images/test_2048_2.bmp", f_width, f_height);
		data[i] = img[i].data;
	}

	NppiSize srcSize = { f_width, f_height };
	NppiRect srcROI = { 0, 0, f_width, f_height };
	NppiRect dstROI = { 0, 0, f_width, f_height };

	for (int i = 0; i < img_num; i++)
	{
		hipMalloc((void**)&d_data[i], sizeof(Npp8u)*f_width*f_height);
		hipMalloc((void**)&d_rotated[i], sizeof(Npp8u)*f_width*f_height);
		hipHostMalloc((void**)&h_result[i], sizeof(Npp8u)*f_width*f_height);
	}

	///////////////////////////*********************** NPP rotation ***********************/////////////////////////
	hipEventRecord(start, 0);
	for (int n = 0; n < n_iter; n++)
	{
		for (int i = 0; i < int(img_num / stream_num); i++)
		{
			for (int j = 0; j < stream_num; j++)
			{
				hipMemcpyAsync(d_data[i*stream_num+j], data[i*stream_num + j], sizeof(Npp8u)*f_width*f_height, hipMemcpyHostToDevice, stream[j]);
				
				// NPP 10.2 and beyond contain an additional element in the NppStreamContext structure
				nppiRotate_8u_C1R_Ctx(d_data[i*stream_num + j], srcSize, f_width, srcROI, d_rotated[i*stream_num + j], f_width, dstROI, angle, 0, 0, NPPI_INTER_LINEAR, nppStreamContext[j]);
				
				hipMemcpyAsync(h_result[i*stream_num + j], d_rotated[i*stream_num + j], sizeof(Npp8u)*f_width*f_height, hipMemcpyDeviceToHost, stream[j]);
			}
		}
	}
	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	printf("Average Rotation Time: %3.1f ms\n", elapsedTime / n_iter);

	for (int i = 0; i < img_num; i++)
	{
		result[i] = Mat(f_height, f_width, CV_8UC1);
		result[i].data = h_result[i];

		sprintf(buf, "output_images/test2048_rotated_nppi_%d.bmp", i);
		imwrite(buf, result[i]);
	}

	return 0;
}

Mat read_BMP_opencv(char* filename, int& w, int& h)
{
	Mat input_img = imread(filename, 0);
	if (input_img.empty())
		throw "Argument Exception";

	int width = input_img.cols;
	int height = input_img.rows;

	w = width;
	h = height;

	return input_img;
}